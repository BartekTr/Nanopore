#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <errno.h>
#include <iostream>
#include <fstream>      // std::filebuf
#include <time.h>

#include <bitset>
#include <list>
#include <map>
#include <set>

#include <thrust/sort.h>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/unique.h>
#include <thrust/device_vector.h>


// TODO For testing
#include <algorithm>


#define K 31
#define MAX_SIZE 1024*1024
#define MIN_K_MER_QUALITY 50
#define AVERAGE_K_MER_QUALITY 80
#define MAX_K_MERS_TO_ALLOCATE 500000000


// Changing DNA code to numbers
__device__ __host__ unsigned long long get_value(char c)
{
    unsigned long long value;
    if (c == 'A')
        value = 0;
    else if (c == 'C')
        value = 1;
    else if (c == 'T')
        value = 2;
    else if (c == 'G')
        value = 3;
    else
        value = 0;

    return value;
}

// Decoding K-mers from numbers to special code
__global__ void SetKMerValues(char* genotype, char* buf, int length, unsigned long long* id_of_good_kmers_GPU, unsigned long long* goodQualityElementsGpu)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < length; i += blockDim.x * gridDim.x)
    {
        unsigned long long decodedValue = 0;
        int quality = 32767;
        int qualitySum = 0;
        for (int k_len = 0; k_len < K; k_len++)
        {
            decodedValue += get_value(genotype[i + k_len]) * (unsigned long long)pow((float)4, (float)K - k_len - 1);

            // Setting K-mer quality as minimum value of single reading
            int currentQuality = (int)buf[i + k_len];
            if (quality > currentQuality)
            {
                quality = currentQuality;
            }

            qualitySum += currentQuality;
        }

        // Using only K-Mers with good quality
        /*if (quality > MIN_K_MER_QUALITY)
        {
            int index = atomicAdd(goodQualityElementsGpu, 1);
            id_of_good_kmers_GPU[index] = decodedValue;
        }*/

        if ((qualitySum / K) > AVERAGE_K_MER_QUALITY)
        {
            int index = atomicAdd(goodQualityElementsGpu, 1);
            id_of_good_kmers_GPU[index] = decodedValue;
        }
    }
}

// Reading encoded K-mer
void print_in_4(unsigned long long value, int k)
{
    for (int i = k-1; i >= 0; --i)
        printf("%lld", (value >> (2 * i)) % 4);
    printf("\n");
}

struct last_mer
{
    const unsigned long long a;

    last_mer(unsigned long long _a) : a(_a) {}

    __host__ __device__
        unsigned long long operator()(const unsigned long long &x) const {
        return x % a;
    }
};

struct first_mer
{
    __host__ __device__
        unsigned long long operator()(const unsigned long long& x) const {
        return x >> 2;
    }
};

int main()
{
    hipSetDevice(0);

    // Timers
    clock_t tStartOfAll = clock();
    clock_t tStartOfAllocationMemory = clock();

    // All K-mers
    long long allElements = 0;
    
    // Temporary data for reading file
    std::filebuf f;
    int length = 0;
    char* buf = (char*)malloc(sizeof(char) * MAX_SIZE);
    char* genotype = (char*)malloc(sizeof(char) * MAX_SIZE);
    char* bufGPU, * genotypeGPU;
    hipMalloc((void**)&bufGPU, sizeof(char) * MAX_SIZE);
    hipMalloc((void**)&genotypeGPU, sizeof(char) * MAX_SIZE);

    // Data for processing file
    unsigned long long elementsWithEnoughQuality = 0;
    unsigned long long* goodQualityElementsGpu;
    hipMalloc((void**)&goodQualityElementsGpu, sizeof(unsigned long long));
    hipMemcpy(goodQualityElementsGpu, &elementsWithEnoughQuality, sizeof(unsigned long long), hipMemcpyHostToDevice);

    // K-Mers with enough quality
    unsigned long long* id_of_all_kmers_GPU;
    hipMalloc((void**)&id_of_all_kmers_GPU, sizeof(unsigned long long) * MAX_K_MERS_TO_ALLOCATE);

    printf("\nTime of allocating memory: %.7fs", (double)(clock() - tStartOfAllocationMemory) / CLOCKS_PER_SEC);
    clock_t tStartOfReading = clock();

    if (f.open("G:/chr100mb.fastq", std::ios::binary | std::ios::in))
    {
        std::istream is(&f);
        int i = 0;
        while (is.getline(buf, MAX_SIZE))
        {
            i++;
            if (i % 4 == 2) // Set A, C, T, G as BYTE
            {
                length = strlen(buf);
                memcpy(genotype, buf, length);
            }

            if (i % 4 == 0) // Set probability
            {
                // Copying currently readed data to Device
                hipMemcpy(bufGPU, buf, sizeof(char) * length, hipMemcpyHostToDevice);
                hipMemcpy(genotypeGPU, genotype, sizeof(char) * length, hipMemcpyHostToDevice);

                // Processing readed data into K-mers with own way
                SetKMerValues << <(length - K)/256 + 1, 256 >> > (genotypeGPU, bufGPU, length - K, id_of_all_kmers_GPU, goodQualityElementsGpu);
                hipDeviceSynchronize();

                allElements += length - K;
            }
        }
    }

    // Delete data for reading file
    free(buf);
    free(genotype);
    hipFree(bufGPU);
    hipFree(genotypeGPU);

    // File close and error handling
    if (f.is_open()) {
        printf("\nFile closed");
        f.close();
    }
    else
    {
        printf("\nError during file closing");
    }

    // Readed data summary
    hipMemcpy(&elementsWithEnoughQuality, goodQualityElementsGpu, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    printf("\nTime of reading: %.7fs", (double)(clock() - tStartOfReading) / CLOCKS_PER_SEC);
    printf("\nAll K-MERS: %llu", allElements);
    printf("\nK-MERS with enough quality: %llu", elementsWithEnoughQuality);
    clock_t tStartOfComputation = clock();
    
    //Sorting K-mers
    thrust::sort(thrust::device, id_of_all_kmers_GPU, id_of_all_kmers_GPU + elementsWithEnoughQuality);
    
    // hashTableLengthv2 - amount of different K-mers
    unsigned long long* hashTableLengthv2;
    hipMalloc((void**)&hashTableLengthv2, sizeof(unsigned long long) * elementsWithEnoughQuality);
    unsigned long long* new_end_for_unique = thrust::unique_copy(thrust::device, id_of_all_kmers_GPU, id_of_all_kmers_GPU + elementsWithEnoughQuality, hashTableLengthv2);
    int hashTableLength = new_end_for_unique - hashTableLengthv2;
    printf("\nUnique K-Mers: %d", hashTableLength);
    hipFree(hashTableLengthv2);

    // Getting amount of different K-mers
    unsigned long long* id_of_kmer_GPU;
    int* amount_of_kmer_GPU;
    hipMalloc((void**)&id_of_kmer_GPU, sizeof(unsigned long long) * hashTableLength);
    hipMalloc((void**)&amount_of_kmer_GPU, sizeof(int) * hashTableLength);
    thrust::pair<unsigned long long*, int*> new_end;
    new_end = thrust::reduce_by_key(thrust::device, id_of_all_kmers_GPU, id_of_all_kmers_GPU + elementsWithEnoughQuality, thrust::make_constant_iterator(1), id_of_kmer_GPU, amount_of_kmer_GPU);
    hipFree(id_of_all_kmers_GPU);

    unsigned long long to_mod = pow(4, K - 1);
    //C array,  weights = amount_of_kmer
    unsigned long long* C;
    hipMalloc((void**)&C, sizeof(unsigned long long) * hashTableLength);
    thrust::transform(thrust::device, id_of_kmer_GPU, id_of_kmer_GPU + hashTableLength, C, last_mer(to_mod));
    unsigned long long* h = (unsigned long long*)malloc(sizeof(unsigned long long) * hashTableLength);
    hipMemcpy(h, C, sizeof(unsigned long long) * hashTableLength, hipMemcpyDeviceToHost);
    hipFree(C);

    //to do R array, transform id_of_kmer_GPU, reduce_by_key and transform again:
    unsigned long long* temp;
    hipMalloc((void**)&temp, sizeof(unsigned long long) * hashTableLength);
    thrust::transform(thrust::device, id_of_kmer_GPU, id_of_kmer_GPU + hashTableLength, temp, first_mer());

    unsigned long long* first;
    int* second;
    hipMalloc((void**)&first, sizeof(unsigned long long) * hashTableLength);
    hipMalloc((void**)&second, sizeof(int) * hashTableLength);
    thrust::pair<unsigned long long*, int*> end;
    end = thrust::reduce_by_key(thrust::device, temp, temp + hashTableLength, thrust::make_constant_iterator(1), first, second);
    hipFree(temp);
    hipFree(first);

    unsigned long long* a = (unsigned long long*)malloc(sizeof(unsigned long long) * hashTableLength);
    unsigned long long* b = (unsigned long long*)malloc(sizeof(unsigned long long) * hashTableLength);
    hipMemcpy(a, second, sizeof(unsigned long long)* hashTableLength, hipMemcpyDeviceToHost);
    hipFree(second);
    b[0] = 0;
    for (int i = 1; i < hashTableLength; i++)
        b[i] = b[i - 1] + a[i - 1];

    free(a);
    free(b);
    free(h);
    hipFree(id_of_kmer_GPU);
    hipFree(amount_of_kmer_GPU);
    hipFree(goodQualityElementsGpu);

    printf("\nTime of computation: %.7fs", (double)(clock() - tStartOfComputation) / CLOCKS_PER_SEC);
    printf("\nAllTime: %.7fs", (double)(clock() - tStartOfAll) / CLOCKS_PER_SEC);

    return 0;
}
